
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void computeHashKernel(const char *input, int len, unsigned int *hash) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < len) {
        unsigned int localHash = 0;
        char c = input[index];
        localHash = (localHash + c) * 31;
        atomicAdd(hash, localHash);
    }
}

extern void computeHash(const char *input, int len, unsigned int &result) {
    char *d_input;
    unsigned int *d_hash;
    unsigned int zero = 0;

    hipMalloc((void **)&d_input, len);
    hipMalloc((void **)&d_hash, sizeof(unsigned int));
    
    hipMemcpy(d_input, input, len, hipMemcpyHostToDevice);
    hipMemcpy(d_hash, &zero, sizeof(unsigned int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (len + blockSize - 1) / blockSize;
    computeHashKernel<<<numBlocks, blockSize>>>(d_input, len, d_hash);

    hipMemcpy(&result, d_hash, sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    result %= 1023;  // Modulo operation as in the CPU version
    std::cout << len;

    hipFree(d_input);
    hipFree(d_hash);
}
