
#include <hip/hip_runtime.h>
#include <iostream>

__device__ unsigned int fnv1_hash(unsigned int previousHash, char c) {
    const unsigned int FNV_prime = 0x01000193; 
    unsigned int hash = previousHash;
    hash *= FNV_prime;
    hash ^= (unsigned int)c;
    return hash;
}

__global__ void computeFNV1HashKernel(const char *input, int len, unsigned int *hash) {
    const unsigned int FNV_offset_basis = 0x811C9DC5; 
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < len) {
        unsigned int localHash = fnv1_hash(FNV_offset_basis, input[index]);
        atomicAdd(hash, localHash);
    }
}

extern void computeFNV1Hash(const char *input, int len, unsigned int &result) {
    char *d_input;
    unsigned int *d_hash;
    unsigned int zero = 0;

    hipMalloc((void **)&d_input, len);
    hipMalloc((void **)&d_hash, sizeof(unsigned int));

    hipMemcpy(d_input, input, len, hipMemcpyHostToDevice);
    hipMemcpy(d_hash, &zero, sizeof(unsigned int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (len + blockSize - 1) / blockSize;
    computeFNV1HashKernel<<<numBlocks, blockSize>>>(d_input, len, d_hash);

    hipMemcpy(&result, d_hash, sizeof(unsigned int), hipMemcpyDeviceToHost);

    result %= 1023;

    hipFree(d_input);
    hipFree(d_hash);
}
