
#include <hip/hip_runtime.h>
#include <iostream>

__device__ unsigned int jenkins_hash_step(unsigned int previousHash, char c) {
    unsigned int hash = previousHash;
    hash += c;
    hash += (hash << 10);
    hash ^= (hash >> 6);
    return hash;
}

__global__ void computeJenkinsHashKernel(const char *input, int len, unsigned int *hash) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < len) {
        unsigned int localHash = jenkins_hash_step(0, input[index]);  // Initialize local hash for each character
        atomicAdd(hash, localHash);
    }
}

extern void computeJenkinsHash(const char *input, int len, unsigned int &result) {
    char *d_input;
    unsigned int *d_hash;
    unsigned int zero = 0;

    hipMalloc((void **)&d_input, len);
    hipMalloc((void **)&d_hash, sizeof(unsigned int));

    hipMemcpy(d_input, input, len, hipMemcpyHostToDevice);
    hipMemcpy(d_hash, &zero, sizeof(unsigned int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (len + blockSize - 1) / blockSize;
    computeJenkinsHashKernel<<<numBlocks, blockSize>>>(d_input, len, d_hash);

    hipMemcpy(&result, d_hash, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Apply final Jenkins mix steps without modulo normalization
    result += (result << 3);
    result ^= (result >> 11);
    result += (result << 15);

    hipFree(d_input);
    hipFree(d_hash);
}
